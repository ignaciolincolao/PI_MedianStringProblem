#include "hip/hip_runtime.h"
#include "kernel.cuh"

__device__ int calculateWeight(int* weights, size_t lenSet, size_t pitchWeights, char x, char y)
{
    int xx = ((int)x-'0'); // transforma el caracter numero a int
    int yy = ((int)y-'0'); // transforma el caracter numero a int
    //printf("sustituci�n: %d %d\n", xx, yy);
    //printf("valor de: %d \n", weights[yy * pitchWeights / sizeof(int) + xx]);
    return weights[xx * pitchWeights / sizeof(int) + yy];
}

__global__ void getDistanceKernel(char* wordsTargets, int* wordsTargetsPos, char* wordSource, int lenWordSource, size_t* pitchMatrix, int** matrix, int* weights, size_t lenSet, size_t pitchWeights, int* distances, int nWords)
{
    //int tidx = blockIdx.x * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int tidx = threadIdx.x + blockDim.x*blockIdx.x;
    
    if (tidx < nWords) {
        //printf("idx %d", tidx);
        int len = wordsTargetsPos[(tidx * 2) + 1] - wordsTargetsPos[tidx * 2]; // largo de la palabra dentro del wordsTargetsPos
        int init = wordsTargetsPos[tidx * 2]; // inicio de la palabra dentro del wordsTargetsPos
        //int end = wordsTargetsPos[(tidx * 2) + 1]; // Fin de la palabra dentro del wordsTargetsPos
        int t1, t2, i, j, costo, res, ancho, min, del, in, sus;
        t1 = len;
        t2 = lenWordSource;


        // Verifica que exista algo que comparar
        if (t1 == 0) return;
        if (t2 == 0) return;
        ancho = pitchMatrix[tidx] / sizeof(int);
        int* m = matrix[tidx];

        // Rellena primera fila y primera columna
        for (i = 0; i <= t2; i++) m[i * ancho + 0] = i * 2;
        for (j = 0; j <= t1; j++) m[j] = j * 2;
        char vWordS, vWordT;
        // Recorremos resto de la matriz llenando pesos
        for (i = 1; i <= t2; i++) for (j = 1; j <= t1; j++)
        {
            vWordS = wordSource[i - 1];
            vWordT = wordsTargets[(init + j) - 1];
            costo = calculateWeight(weights, lenSet, pitchWeights, vWordS, vWordT);
            // Calcula el minimo 
            del = m[i * ancho + j - 1] + 2; // El mas dos es el costo de borrado
            in = m[(i - 1) * ancho + j] + 2; // El mas dos es el costo de inserci�n
            sus = m[(i - 1) * ancho + j - 1] + costo; // Costo de sustituci�n
            min = del < in ? del : in;
            min = min < sus ? min : sus;
            // guarda en la matriz el minimo
            m[i * ancho + j] = min;
        }

        res = m[t2 * ancho + t1];

        distances[tidx] = res;
    }
    return;

}

__global__ void levenshteinKernel(char* wordsTargets, int* wordsTargetsPos, char* wordSource, int lenWordSource, size_t* pitchMatrix, int** matrix, int** actions, int* weights, size_t lenSet, size_t pitchWeights, int nWords)
{
    //int tidx = blockIdx.x * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int tidx = threadIdx.x + blockDim.x * blockIdx.x;
    if (tidx < nWords) {
        int len = wordsTargetsPos[(tidx * 2) + 1] - wordsTargetsPos[tidx * 2]; // largo de la palabra dentro del wordsTargetsPos
        int init = wordsTargetsPos[tidx * 2]; // inicio de la palabra dentro del wordsTargetsPos
        int  i, j, costo, min, del, in, sus;
        const int t1 = len;
        const int t2 = lenWordSource;
        //printf("%d | %d |%d | %d | %d \n", t1, t2, init, end, len);
        for (i = 0; i <= t2; i++) 

        // Verifica que exista algo que comparar
        if (t1 == 0) return;
        if (t2 == 0) return;
        //ancho = pitchMatrix[tidx] / sizeof(int);
        const int ancho = pitchMatrix[tidx] / sizeof(int);
        //printf("pitch %d\n", pitchMatrix[tidx]);
        int* m = matrix[tidx];

        // Rellena primera fila y primera columna
        for (i = 0; i <= t2; i++) m[i * ancho + 0] = i * 2;
        for (j = 0; j <= t1; j++) {
            m[j] = j * 2;
        }
        // Recorremos resto de la matriz llenando pesos
        char vWordS, vWordT;
        for (i = 1; i <= t2; i++) for (j = 1; j <= t1; j++)
        {
            vWordS = wordSource[i - 1];
            vWordT = wordsTargets[(init + j) - 1];
            //printf("%d | %d \n", i, j);
            //printf("%c | %c \n", wordSource[i - 1], wordsTargets[(init + j) - 1]);
            costo = calculateWeight(weights, lenSet, pitchWeights, vWordS, vWordT);
            // Calcula el minimo 
            del = m[i * ancho + j - 1] + 2; // El mas dos es el costo de borrado
            in = m[(i - 1) * ancho + j] + 2; // El mas dos es el costo de inserci�n
            sus = m[(i - 1) * ancho + j - 1] + costo; // Costo de sustituci�n
            min = del < in ? del : in;
            min = min < sus ? min : sus;
            // guarda en la matriz el minimo
            m[i * ancho + j] = min;
        }
        
        int del_t, del_w, ins_t, ins_w, sus_t, sus_w, x, y, count,cDis, postarget, posSource;
        int* listActions = actions[tidx];
        x = t1;
        y = t2;
        count = 1;
        listActions[0] = min;
        //if (tidx == 0) {
        
        while (x != 0 || y != 0)
        {
            postarget = x - 1 < 0 ? 0 : x - 1; // La palabra dinamica
            posSource = y - 1 < 0 ? 0 : y - 1; // La palabra constante
            vWordS = wordSource[y - 1];
            vWordT = wordsTargets[(init + x) - 1];

            del_w = (y > 0) ? 2 : INT_MAX;
            del_t = (y > 0) ? m[posSource * ancho + x] + del_w : INT_MAX; // Borrado

            ins_w = (x > 0) ? 2 : INT_MAX;
            ins_t = (x > 0) ? m[y * ancho + postarget] + ins_w : INT_MAX; // inserrci�n

            sus_w = (x > 0 && y > 0) ? calculateWeight(weights, lenSet, pitchWeights, vWordS, vWordT) : INT_MAX;
            sus_t = (x > 0 && y > 0) ? m[posSource * ancho + postarget] + sus_w : INT_MAX;

            cDis = m[y * ancho + x];
            if (sus_t == cDis)
            {
                if (vWordS != vWordT) {
                    listActions[count] = posSource;
                    listActions[count + 1] = 1;
                    listActions[count + 2] = ((int)vWordT - '0');
                  
                    count += 3;
                }
                y--;
                x--;
            }
            else if (ins_t == cDis)
            {
                listActions[count] = posSource;
                listActions[count + 1] = 0;
                listActions[count + 2] = ((int)vWordT - '0');
                x--;
                count += 3;
            }
            else if (del_t == cDis)
            {
                listActions[count] = posSource;
                listActions[count + 1] = 2;
                listActions[count + 2] = NULL;
                y--;
                count += 3;
            }
            else
            {
                printf("error en la lista de operaciones");
                break;
            }

        }
        listActions[count] = -1;
        
    }

}
